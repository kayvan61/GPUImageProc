#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>

#include "Image.hpp"

#define COLOR_VALS 255
#define ONE_HIST_SIZE 255 * 3
#define BLOCKSIZE 1024

__global__ void reduce(int *histOut, int* localHistsIn) {
    unsigned res_idx = threadIdx.x;

    unsigned histogramIndex = (blockIdx.y * blockDim.x * ONE_HIST_SIZE) + (blockIdx.x * ONE_HIST_SIZE);
    unsigned histogramIndex_r = histogramIndex + res_idx;
    unsigned histogramIndex_g = histogramIndex + COLOR_VALS + res_idx;
    unsigned histogramIndex_b = histogramIndex + (2*COLOR_VALS) + res_idx;

    int val = localHistsIn[histogramIndex_r];

    atomicAdd(&(histOut[res_idx]), val);
}

__global__ void histogram(int* localHistOut, Pixel **img_in, unsigned image_w, unsigned image_h) {
    unsigned img_x = blockIdx.y * BLOCKSIZE + threadIdx.x;
    unsigned img_y = blockIdx.y;

    if(img_y >= image_h || img_x >= image_w) {
        return;
    }

    unsigned histogramIndex = (blockIdx.y * blockDim.x * ONE_HIST_SIZE) + (blockIdx.x * ONE_HIST_SIZE);
    unsigned histogramIndex_r = histogramIndex + img_in[img_y][img_x].r;
    unsigned histogramIndex_g = histogramIndex + COLOR_VALS + img_in[img_y][img_x].g;
    unsigned histogramIndex_b = histogramIndex + (2*COLOR_VALS) + img_in[img_y][img_x].b;

    atomicAdd(&(localHistOut[histogramIndex_r]), 1);
    atomicAdd(&(localHistOut[histogramIndex_g]), 1);
    atomicAdd(&(localHistOut[histogramIndex_b]), 1);
}

template<typename T>
void cuda2dAlloc(T*** outBuffer, unsigned w, unsigned h) {
    hipMalloc(outBuffer, sizeof(T*) * h);
    T **temp = (T **)malloc(sizeof(T*) * h);
    for(unsigned i = 0; i < h; i++) {
        hipMalloc((void**)(&temp[i]), sizeof(T) * w);
    }
    hipMemcpy(*outBuffer, temp, sizeof(T*) * h, ::hipMemcpyHostToDevice);
    free(temp);
}

int main(int argc, char** argv) {
    if(argc != 2) {
	  std::cout << "use: ./main <image in>" << std::endl;
	}

    Image img(argv[1]);
    
    std::pair<unsigned, unsigned> imgDims;
    imgDims = img.getImageDims();
    unsigned w = imgDims.first;
    unsigned h = imgDims.second;
    printf("image dims: %d %d\n", w, h);

    dim3 pictureTBDim = {BLOCKSIZE, 1};
    dim3 blocksTBDim = {(w/BLOCKSIZE) + 1, h};

    int* devHist;
    int* devLocalHists;
    hipMalloc((void**)&devLocalHists, sizeof(int) * blocksTBDim.x * blocksTBDim.y * ONE_HIST_SIZE);
    hipMalloc((void**)&devHist, sizeof(int) * ONE_HIST_SIZE);
    hipMemset(devHist, 0, sizeof(int) * ONE_HIST_SIZE);

    img.copyToDevice();

    printf("thread dims: x: %d y: %d\n", pictureTBDim.x, pictureTBDim.y);
    printf("block  dims: x: %d y: %d\n", blocksTBDim.x, blocksTBDim.y);
    histogram<<<blocksTBDim,pictureTBDim>>>(devLocalHists, img.getRawDeviceBuffer(), w, h);
    reduce<<<blocksTBDim, COLOR_VALS>>>(devHist, devLocalHists);

    hipDeviceSynchronize();

    int *host = (int*)malloc(sizeof(int) * COLOR_VALS);
    hipMemcpy(host, devHist, sizeof(int) * COLOR_VALS, hipMemcpyDeviceToHost);
    for(int i = 0; i < COLOR_VALS; i++) {
        printf("%d ", host[i]);
    }
    printf("\n");

    return 0;
}
