#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>

#include "Image.hpp"

#define COLOR_VALS 256
#define ONE_HIST_SIZE  COLOR_VALS * 3
#define BLOCKSIZE 1024

__global__ void reduce(int *histOut, int* localHistsIn) {
    unsigned res_idx = threadIdx.x;

    unsigned histogramIndex = (blockIdx.y * gridDim.x * ONE_HIST_SIZE) + (blockIdx.x * ONE_HIST_SIZE) + res_idx;

    int val = localHistsIn[histogramIndex];

    atomicAdd(&(histOut[res_idx]), val);
}

__global__ void histogram(int* localHistOut, Pixel *img_in, unsigned image_w, unsigned image_h) {
    unsigned img_x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned img_y = blockIdx.y;

    if(img_y >= image_h || img_x >= image_w) {
        return;
    }

    unsigned histogramIndex = (blockIdx.y * gridDim.x + blockIdx.x) * ONE_HIST_SIZE;
    unsigned histogramIndex_r = histogramIndex + (img_in[img_y * image_w + img_x].r * 3);
    unsigned histogramIndex_g = histogramIndex + (img_in[img_y * image_w + img_x].g * 3)+1;
    unsigned histogramIndex_b = histogramIndex + (img_in[img_y * image_w + img_x].b * 3)+2;

    atomicAdd(&(localHistOut[histogramIndex_r]), 1);
    atomicAdd(&(localHistOut[histogramIndex_g]), 1);
    atomicAdd(&(localHistOut[histogramIndex_b]), 1);

}

int main(int argc, char** argv) {
    if(argc != 2) {
	  std::cout << "use: ./main <image in>" << std::endl;
	}

    Image img(argv[1]);
    
    std::pair<unsigned, unsigned> imgDims;
    imgDims = img.getImageDims();
    unsigned w = imgDims.first;
    unsigned h = imgDims.second;
    printf("image dims: %d %d\n", w, h);

    dim3 pictureTBDim = {BLOCKSIZE, 1};
    dim3 blocksTBDim = {(w/BLOCKSIZE) + 1, h};

    int* devHist;
    int* devLocalHists;
    hipMalloc((void**)&devLocalHists, sizeof(int) * blocksTBDim.x * blocksTBDim.y * ONE_HIST_SIZE);
    hipMemset(devLocalHists, 0, sizeof(int) * blocksTBDim.x * blocksTBDim.y * ONE_HIST_SIZE);
    hipMalloc((void**)&devHist, sizeof(int) * ONE_HIST_SIZE);
    hipMemset(devHist, 0, sizeof(int) * ONE_HIST_SIZE);

    img.copyToDevice();

    printf("thread dims: x: %d y: %d\n", pictureTBDim.x, pictureTBDim.y);
    printf("block  dims: x: %d y: %d\n", blocksTBDim.x, blocksTBDim.y);
    histogram<<<blocksTBDim,pictureTBDim>>>(devLocalHists, img.getRawDeviceBuffer(), w, h);
    reduce<<<blocksTBDim, ONE_HIST_SIZE>>>(devHist, devLocalHists);

    hipDeviceSynchronize();

    int *host = (int*)malloc(sizeof(int) * ONE_HIST_SIZE);
    hipMemcpy(host, devHist, sizeof(int) * ONE_HIST_SIZE, hipMemcpyDeviceToHost);
    int sum = 0;
    printf("====== result ======\n");
    for(int i = 0; i < ONE_HIST_SIZE; i++) {
        sum += host[i];
        printf("%d ", host[i]);
    }
    printf("\n");
    printf("sum: %d\n", sum);

    return 0;
}
