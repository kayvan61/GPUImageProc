#include "hip/hip_runtime.h"
#include <iostream>

__global__ void populate(int *a) {
    printf("%d, %d, %d, %d, %d, %d\n", blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);
    int idx = blockIdx.x * 32 + threadIdx.x;
    a[idx] = idx;
}

int main() {
    int *retVal;
    int *a;
    retVal = (int*)malloc(sizeof(int) * 32 * 512);
    hipMalloc((void**)&a, sizeof(int) * 32 * 512);
    dim3 threadBlockDim = {32};
    dim3 BlockDim = {512};
    populate<<<BlockDim, threadBlockDim>>>(a);
    hipMemcpy(retVal, a, sizeof(int) * 32 * 512, ::hipMemcpyDeviceToHost);
    for(int i = 512; i < 512 + 32; i++) {
        std::cout << retVal[i] << " ";
    }
    std::cout << "\n";
    return 0;
}